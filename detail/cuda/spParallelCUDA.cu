#include "hip/hip_runtime.h"
//
// Created by salmon on 16-7-25.
//

#include </usr/local/cuda/include/hip/hip_runtime_api.h>
#include "../../../../../../../usr/local/cuda/include/"

extern "C" {
#include <assert.h>
#include "spParallelCUDA.h"
#include "../../spMPI.h"
}


int spParallelDeviceInitialize(int argc, char **argv)
{
    int num_of_device = 0;
    SP_DEVICE_CALL(hipGetDeviceCount(&num_of_device));
    SP_DEVICE_CALL(hipSetDevice(spMPIRank() % num_of_device));
    SP_DEVICE_CALL(hipDeviceSynchronize()); // Wait for the GPU launched work to complete
    SP_DEVICE_CALL(hipGetLastError());
//    SP_DEVICE_CALL(hipDeviceSetCacheConfig(hipFuncCachePreferL1));
    return SP_SUCCESS;
}

int spParallelDeviceFinalize()
{
    SP_DEVICE_CALL(hipDeviceReset());
    return SP_SUCCESS;
}

int spMemoryDeviceAlloc(void **p, size_type s)
{
    if (s > 0) {SP_DEVICE_CALL(hipMalloc(p, s)); }
    else { *p = NULL; }
    return SP_SUCCESS;
}

int spMemoryDeviceFree(void **_P_)
{
    if (*_P_ != NULL)
    {
        SP_DEVICE_CALL(hipFree(*_P_));
        *_P_ = NULL;
    }
    return SP_SUCCESS;;
};

int spMemoryCopy(void *dest, void const *src, size_type s)
{
    SP_DEVICE_CALL(hipMemcpy(dest, src, s, hipMemcpyDefault));
    return SP_SUCCESS;
}

int spMemoryCopyToCache(const void *dest, void const *src, size_type s)
{
    SP_DEVICE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dest), src, s));
    return SP_SUCCESS;
}

int spMemorySet(void *dest, int v, size_type s)
{
    SP_DEVICE_CALL (hipMemset(dest, v, s));
    return SP_SUCCESS;
}

int spMemoryHostAlloc(void **p, size_type s)
{
    SP_DEVICE_CALL (hipHostAlloc(p, s, hipHostMallocDefault));
    return SP_SUCCESS;

};

int spMemoryHostFree(void **p)
{
    if (*p != NULL)
    {
        SP_DEVICE_CALL(hipHostFree(*p));
        *p = NULL;
    }
    return SP_SUCCESS;
}


__global__
void spParallelDeviceFillIntKernel(int *d, int v, size_type max)
{
    for (size_t s = threadIdx.x + blockIdx.x * blockDim.x; s < max; s += gridDim.x * blockDim.x) { d[s] = v; }
};

int spParallelDeviceFillInt(int *d, int v, size_type s)
{

    SP_CALL_DEVICE_KERNEL(spParallelDeviceFillIntKernel, 16, 256, d, v, s);

    return SP_SUCCESS;
};

__global__
void spParallelDeviceFillRealKernel(Real *d, Real v, size_type max)
{
    for (size_type s = threadIdx.x + blockIdx.x * blockDim.x; s < max; s += gridDim.x * blockDim.x) { d[s] = v; }
};

int spParallelDeviceFillReal(Real *d, Real v, size_type s)
{
    SP_CALL_DEVICE_KERNEL(spParallelDeviceFillRealKernel, 16, 256, d, v, s);
    return SP_SUCCESS;
};


__global__
void spParallelAssignKernel(size_type max, size_type const *offset, Real *d, Real const *v)
{

    size_type num_of_thread = blockDim.x * gridDim.x * blockDim.x * gridDim.x * blockDim.x * gridDim.x;

    for (size_type s = (threadIdx.x + blockIdx.x * blockDim.x) +
                       (threadIdx.y + blockIdx.y * blockDim.y) * blockDim.x * gridDim.x +
                       (threadIdx.x + blockIdx.x * blockDim.x) * blockDim.x * gridDim.x * blockDim.y * gridDim.y;
         s < max; s += num_of_thread) { d[offset[s]] = v[s]; }
};

int spParallelAssign(size_type num_of_point, size_type *offset, Real *d, Real const *v)
{
    SP_CALL_DEVICE_KERNEL(spParallelAssignKernel, 16, 256, num_of_point, offset, d, v);
    return SP_SUCCESS;
};
//
//int spMemoryDeviceToHost(void **p, void *src, size_type size_in_byte)
//{
//    int  SP_SUCCESS = SP_SUCCESS;
//    SP_CALL(spMemoryHostAlloc(p, size_in_byte));
//    SP_CALL(spMemoryCopy(*p, src, size_in_byte));
//    return  SP_SUCCESS;
//
//}
//
//int spMemoryHostFree(void **p)
//{
//    int  SP_SUCCESS = SP_SUCCESS;
//    SP_CALL(spMemoryHostFree(p));
//    return  SP_SUCCESS;
//
//}