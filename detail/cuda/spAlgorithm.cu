#include "hip/hip_runtime.h"
//
// Created by salmon on 16-9-13.
//
extern "C"
{
#include <assert.h>
#include "../../spAlogorithm.h"
#include "../sp_device.h"
#include "../../spDataType.h"
#include "../../spMisc.h"
}

#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include <thrust/scan.h>

#include "../../../../../../../usr/local/cuda/include/host_defines.h"
#include "../../../../../../../usr/local/cuda/include/"


int sort_by_key(size_type const *key_start, size_type const *key_end, size_type *value)
{
    thrust::sort_by_key(thrust::device_ptr<size_type>((size_type *) key_start),
                        thrust::device_ptr<size_type>((size_type *) key_end),
                        thrust::device_ptr<size_type>(value));

    return SP_SUCCESS;
};

__global__
void spMemoryIndirectCopyKernel(Real *dest, Real const *src, size_type num, size_type const *index)
{
    uint s = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    if (s < num) { dest[s] = src[index[s]]; }
};

int spMemoryCopyIndirect(Real *dest, Real const *src, size_type num, size_type const *index)
{

    if (num > 0)
    {
        SP_CALL_DEVICE_KERNEL(spMemoryIndirectCopyKernel, num / NUMBER_OF_THREADS_PER_BLOCK + 1,
                              NUMBER_OF_THREADS_PER_BLOCK, dest, src, num, index);
    }
    return SP_SUCCESS;
}


__global__
void spMemoryInvIndirectCopyKernel(Real *dest, Real const *src, size_type num, size_type const *index)
{
    uint s = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    if (s < num) { dest[index[s]] = src[s]; }
};

int spMemoryCopyInvIndirect(Real *dest, Real const *src, size_type num, size_type const *index)
{
    if (num > 0) { SP_CALL_DEVICE_KERNEL(spMemoryInvIndirectCopyKernel, num / 256 + 1, 256, dest, src, num, index); }
    return SP_SUCCESS;
}

__global__
void spMemoryCopySubArrayRealKernel(Real *dest, Real const *src,
                                    dim3 strides,
                                    dim3 start,
                                    dim3 count)
{
    uint x = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    uint y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
    uint z = __umul24(blockIdx.z, blockDim.z) + threadIdx.z;

    if (x < count.x && y < count.y && z < count.z)
    {
        dest[(x * count.y + y) * count.z + z] = src[(start.x + x) * strides.x +
                                                    (start.y + y) * strides.y +
                                                    (start.z + z) * strides.z];


    }

}

__global__
void spMemoryCopySubArraySizeTypeKernel(size_type *dest, size_type const *src,
                                        dim3 strides,
                                        dim3 start,
                                        dim3 count)
{
    uint x = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    uint y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
    uint z = __umul24(blockIdx.z, blockDim.z) + threadIdx.z;

    if (x < count.x && y < count.y && z < count.z)
    {
        dest[(x * count.y + y) * count.z + z] = src[(start.x + x) * strides.x +
                                                    (start.y + y) * strides.y +
                                                    (start.z + z) * strides.z];


    }

}

int spMemoryCopySubArray(void *dest, void const *src, int type_tag, size_type const *strides, size_type const *start,
                         size_type const *count)
{
    if (count[0] * count[1] * count[2] == 0) { return SP_SUCCESS; }

    size_type grid_dim[3] = {count[0], count[1], count[2]};

    size_type block_dim[3] = {1, 1, 1};

    SP_CALL(spParallelThreadBlockDecompose(128, grid_dim, block_dim));


    switch (type_tag)
    {
        case SP_TYPE_size_type:
        SP_CALL_DEVICE_KERNEL(spMemoryCopySubArraySizeTypeKernel, sizeType2Dim3(grid_dim), sizeType2Dim3(block_dim),
                              (size_type *) dest, (size_type const *) src, sizeType2Dim3(strides), sizeType2Dim3(start),
                              sizeType2Dim3(count));
            break;


        case SP_TYPE_Real:
        SP_CALL_DEVICE_KERNEL(spMemoryCopySubArrayRealKernel, sizeType2Dim3(grid_dim), sizeType2Dim3(block_dim),
                              (Real *) dest, (Real const *) src, sizeType2Dim3(strides), sizeType2Dim3(start),
                              sizeType2Dim3(count));
            break;

        default:
            UNIMPLEMENTED;
            break;
    }
//    SP_CALL(printArray(dest, type_tag, 3, count));

    return SP_SUCCESS;
}

__global__
void spMemoryCopyInvSubArrayRealKernel(Real *dest, Real const *src, dim3 strides, dim3 start, dim3 count)
{
    uint x = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    uint y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
    uint z = __umul24(blockIdx.z, blockDim.z) + threadIdx.z;

    if (x < count.x && y < count.y && z < count.z)
    {
        dest[(start.x + x) * strides.x +
             (start.y + y) * strides.y +
             (start.z + z) * strides.z] = src[(x * count.y + y) * count.z + z];
    }

}

__global__
void spMemoryCopyInvSubArraySizeTypeKernel(size_type *dest, size_type const *src, dim3 strides, dim3 start, dim3 count)
{
    uint x = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    uint y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
    uint z = __umul24(blockIdx.z, blockDim.z) + threadIdx.z;

    if (x < count.x && y < count.y && z < count.z)
    {
        dest[(start.x + x) * strides.x +
             (start.y + y) * strides.y +
             (start.z + z) * strides.z] = src[(x * count.y + y) * count.z + z];
    }

}

int spMemoryCopyInvSubArray(void *dest, void const *src, int type_tag, size_type const *strides, size_type const *start,
                            size_type const *count)
{
    if (count[0] * count[1] * count[2] == 0) { return SP_SUCCESS; }
    size_type grid_dim[3] = {count[0], count[1], count[2]};
    size_type block_dim[3];

    SP_CALL(spParallelThreadBlockDecompose(128, grid_dim, block_dim));

    switch (type_tag)
    {


        case SP_TYPE_size_type:
        SP_CALL_DEVICE_KERNEL(spMemoryCopyInvSubArraySizeTypeKernel, sizeType2Dim3(grid_dim), sizeType2Dim3(block_dim),
                              (size_type *) dest, (size_type const *) src, sizeType2Dim3(strides), sizeType2Dim3(start),
                              sizeType2Dim3(count));
            break;


        case SP_TYPE_Real:
        SP_CALL_DEVICE_KERNEL(spMemoryCopyInvSubArrayRealKernel, sizeType2Dim3(grid_dim), sizeType2Dim3(block_dim),
                              (Real *) dest, (Real const *) src, sizeType2Dim3(strides), sizeType2Dim3(start),
                              sizeType2Dim3(count));

            break;

        default:
            UNIMPLEMENTED;
            break;


    }
    return SP_SUCCESS;


}

__global__
void spFillSeqRealKernel(Real *v, size_type num, size_type min, size_type step)
{
    uint s = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    if (s < num) { v[s] = s * step + min; }
};
__global__
void spFillSeqSizeTypeKernel(size_type *v, size_type num, size_type min, size_type step)
{
    uint s = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    if (s < num) { v[s] = s * step + min; }
};

int spFillSeq(void *v, int type_tag, size_type num, size_type min, size_type step)
{
    switch (type_tag)
    {
        case SP_TYPE_Real:
        SP_CALL_DEVICE_KERNEL(spFillSeqRealKernel, num / NUMBER_OF_THREADS_PER_BLOCK + 1, NUMBER_OF_THREADS_PER_BLOCK,
                              (Real *) v, num, min, step);
            break;
        case SP_TYPE_size_type:
        SP_CALL_DEVICE_KERNEL(spFillSeqSizeTypeKernel,
                              num / NUMBER_OF_THREADS_PER_BLOCK + 1, NUMBER_OF_THREADS_PER_BLOCK,
                              (size_type *) v, num, min, step);
            break;
        default:
            UNIMPLEMENTED;
            break;

    }
    return SP_SUCCESS;
};

__global__
void spTransformMinusKernel(size_type *v, size_type const *a, size_type const *b, size_type num)
{
    uint s = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    if (s < num) { v[s] = a[s] - b[s]; }
};

int spTransformMinus(size_type *v, size_type const *a, size_type const *b, size_type num)
{
    /*@formatter:off*/
   spTransformMinusKernel<<<num/256+1,256>>>(v,a,b,num);
    /*@formatter:on*/
    return SP_SUCCESS;

};
__global__
void spTransformAddKernel(size_type *v, size_type const *a, size_type const *b, size_type num)
{
    uint s = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    if (s < num) { v[s] = a[s] + b[s]; }
};

int spTransformAdd(size_type *v, size_type const *a, size_type const *b, size_type num)
{
    /*@formatter:off*/
   spTransformAddKernel<<<num/256+1,256>>>(v,a,b,num);
    /*@formatter:on*/
    return SP_SUCCESS;

}

int spExclusiveScan(size_type const *b, size_type const *e, size_type *out)
{
    try
    {
        thrust::exclusive_scan(
                thrust::device_ptr<size_type>((size_type *) b),
                thrust::device_ptr<size_type>((size_type *) e),
                thrust::device_ptr<size_type>(out)
        );
    }
    catch (...)
    {
        return SP_FAILED;
    }

    return SP_SUCCESS;
};

int spInclusiveScan(size_type const *b, size_type const *e, size_type *out)
{
    try
    {
        thrust::inclusive_scan(
                thrust::device_ptr<size_type>((size_type *) b),
                thrust::device_ptr<size_type>((size_type *) e),
                thrust::device_ptr<size_type>(out)
        );
    }
    catch (...)
    {
        return SP_FAILED;
    }


    return SP_SUCCESS;
};


__global__ void
spPackInt_kernel(size_type *dest, size_type const *src,
                 size_type const *dest_start, size_type const *src_start, size_type const *count)
{

    uint s = threadIdx.x;
    while (s < count[blockIdx.x])
    {
        dest[dest_start[blockIdx.x] + threadIdx.x] = src[src_start[blockIdx.x] + threadIdx.x];
        s += blockDim.x;
    }

}

int spPackInt(size_type **dest, size_type *num, size_type const *src,
              size_type num_of_cell, size_type const *start, size_type const *count)
{
    if (num_of_cell == 0) { return SP_SUCCESS; }

    size_type * dest_start;

    SP_CALL(spMemoryDeviceAlloc((void **) &dest_start, (num_of_cell + 1) * sizeof(size_type)));

    SP_CALL(spInclusiveScan(count, count + num_of_cell, dest_start + 1));

    SP_CALL(spMemoryCopy(num, &(dest_start[num_of_cell]), sizeof(size_type)));

    SP_CALL(spMemorySet(dest_start, 0, sizeof(size_type)));

    SP_CALL(spMemoryDeviceAlloc((void **) dest, (*num) * sizeof(size_type)));

    SP_CALL_DEVICE_KERNEL(spPackInt_kernel, num_of_cell, 256, *dest, src, dest_start, start, count);

    SP_CALL(spMemoryDeviceFree((void **) &dest_start));

    return SP_SUCCESS;
}